
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void add( int a, int b, int *c )
{ 
	*c = a + b; 
} 
int main (void)
{
	int sum;
	int *sum_dev;
	hipMalloc((void**)&sum_dev, sizeof(int));
	add<<<1,1>>>(1,10,sum_dev);	  
	hipMemcpy(&sum,sum_dev,sizeof(int),hipMemcpyDeviceToHost);
	printf("Addition of 1 + 10 = %d\n",sum);
	hipFree(sum_dev);
}